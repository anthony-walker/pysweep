#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/device_functions.h>

__device__ int *boyo;

__global__ void gimme()
{
    printf("%d \n", boyo[threadIdx.x]);
}

int main(int argc, char *argv[])
{
    int mb[32];
    hipMalloc((void **) &boyo, 32*4);
    for (int i=0; i<32; i++)
    {
        mb[i] = i*i;
    }
    hipMemcpy(boyo, &mb, 32*4, hipMemcpyHostToDevice);
    gimme <<<1, 32>>> ();

    hipFree(boyo);
    return 0;

}
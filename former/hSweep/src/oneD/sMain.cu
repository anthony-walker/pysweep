#include "hip/hip_runtime.h"
/*
    Entry point for hsweep.
*/

#include <fstream>

#include "cudaUtils.h"
#include "heads.h"
#include "decomp.h"
#include "classic.h"
#include "swept.h"

/**
----------------------
    MAIN PART
----------------------
*/

int main(int argc, char *argv[])
{
    makeMPI(argc, argv);

    if (!ranks[1]) cudaRunCheck();

    #ifdef NOS
        if (!ranks[1]) std::cout << "No Solution Version." << std::endl;
    #endif

    std::string i_ext = ".json";
    std::string t_ext = ".csv";
    std::string myrank = std::to_string(ranks[1]);
    std::string scheme = argv[1];

    // Equation, grid, affinity data
    std::ifstream injson(argv[2], std::ifstream::in);
    injson >> inJ;
    injson.close();

    parseArgs(argc, argv);
    initArgs();

    int prevGpu=0; //Get the number of GPUs in front of the current process.
    int gpuPlaces[nprocs]; //Array of 1 or 0 for number of GPUs assigned to process

    //If there are no GPUs or if the GPU Affinity is 0, this block is unnecessary.
    if (cGlob.nGpu > 0)
    {
        MPI_Allgather(&cGlob.hasGpu, 1, MPI_INT, &gpuPlaces[0], 1, MPI_INT, MPI_COMM_WORLD);
        for (int k=0; k<ranks[1]; k++) prevGpu+=gpuPlaces[k];
    }

    cGlob.xStart = cGlob.xcpu * ranks[1] + cGlob.xg * prevGpu;
    states **state;

    int exSpace = ((int)!scheme.compare("S") * cGlob.ht) + 2;
    int xc = (cGlob.hasGpu) ? cGlob.xcpu/2 : cGlob.xcpu;
    int nrows = (cGlob.hasGpu) ? 3 : 1;
    int xalloc = xc + exSpace;

    std::string pth = string(argv[3]);


    if (cGlob.hasGpu)
    {
        state = new states* [3];
        hipHostAlloc((void **) &state[0], xalloc * cGlob.szState, hipHostMallocDefault);
        hipHostAlloc((void **) &state[1], (cGlob.xg + exSpace) * cGlob.szState, hipHostMallocDefault);
        hipHostAlloc((void **) &state[2], xalloc * cGlob.szState, hipHostMallocDefault);

        cout << "Rank: " << ranks[1] << " has a GPU" << endl;
        int ii[3] = {xc, cGlob.xg, xc};
        int xi;
        for (int i=0; i<3; i++)
        {
            xi = cGlob.xStart-1;
            for (int n=0; n<i; n++) xi += ii[n];
            for (int k=0; k<(ii[i]+2); k++)  initialState(inJ, state[i], k, xi);
        }

        hipMemcpyToSymbol(HIP_SYMBOL(deqConsts), &heqConsts, sizeof(eqConsts));

//        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    }
    else
    {
        state = new states*[1];
        state[0] = new states[xalloc * cGlob.szState];
        for (int k=0; k<(xc+2); k++)  initialState(inJ, state[0], k, cGlob.xStart-1);
    }

    writeOut(state, 0.0);

    // If you have selected scheme I, it will only initialize and output the initial values.

    if (scheme.compare("I"))
    {
        int tstep = 1;
        double timed, tfm;

		if (!ranks[1])
		{
            printf ("Scheme: %s - Grid Size: %d - Affinity: %.2f\n", scheme.c_str(), cGlob.nX, cGlob.gpuA);
            printf ("threads/blk: %d - timesteps: %.2f\n", cGlob.tpb, cGlob.tf/cGlob.dt);
		}

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = MPI_Wtime();

        if (!scheme.compare("C"))
        {
            tfm = classicWrapper(state, &tstep);
        }
        else if  (!scheme.compare("S"))
        {
            tfm = sweptWrapper(state, &tstep);
        }
        else
        {
            std::cerr << "Incorrect or no scheme given" << std::endl;
        }

        MPI_Barrier(MPI_COMM_WORLD);
        if (!ranks[1]) timed = (MPI_Wtime() - timed);

        if (cGlob.hasGpu)
		{
			hipError_t error = hipGetLastError();
        	if(error != hipSuccess)
        	{
            	// print the CUDA error message and exit
            	printf("CUDA error: %s\n", hipGetErrorString(error));
            	exit(-1);
        	}
			hipDeviceSynchronize();
		}

        writeOut(state, tfm);

        if (!ranks[1])
        {
            timed *= 1.e6;

            double n_timesteps = tfm/cGlob.dt;

            double per_ts = timed/n_timesteps;

            std::cout << n_timesteps << " timesteps" << std::endl;
            std::cout << "Averaged " << per_ts << " microseconds (us) per timestep" << std::endl;

            // Write out performance data as csv
            std::string tpath = pth + "/t" + fspec + scheme + t_ext;
            FILE * timeOut;
            timeOut = fopen(tpath.c_str(), "a+");
            fseek(timeOut, 0, SEEK_END);
            int ft = ftell(timeOut);
            if (!ft) fprintf(timeOut, "tpb,gpuA,nX,time\n");
            fprintf(timeOut, "%d,%.4f,%d,%.8f\n", cGlob.tpb, cGlob.gpuA, cGlob.nX, per_ts);
            fclose(timeOut);
        }
    }
        //WRITE OUT JSON solution to differential equation

	#ifndef NOS
        std::string spath = pth + "/s" + fspec + "_" + myrank + i_ext;
        std::ofstream soljson(spath.c_str(), std::ofstream::trunc);
        if (!ranks[1]) solution["meta"] = inJ;
        soljson << solution;
        soljson.close();
	#endif

    if (cGlob.hasGpu)
    {
        for (int k=0; k<3; k++) hipHostFree(state[k]);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
    else
    {
        delete[] state[0];
    }
    delete[] state;

    endMPI();
    return 0;
}

//inline void cudaCheck(hipError_t code, const char *file, int line, bool abort=false)
//{
//   if (code != hipSuccess)
//   {
//      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
//      if (abort) exit(code);
//   }
//}
